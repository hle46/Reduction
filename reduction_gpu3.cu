#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdlib>

#define BLOCK_SIZE 128
#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

using namespace std;

__global__ void getMin(float *input, int *input_idx, int offset, int len, int offset_out, float *output_val,
                       int *output_idx) {
  __shared__ float smem_val[BLOCK_SIZE];
  __shared__ int smem_idx[BLOCK_SIZE];

  int tx = threadIdx.x;

  int i = tx + blockIdx.x * BLOCK_SIZE * 8 + offset;

  float min_val = INFINITY;
  int min_idx = i;

  if (i < len) {
    float a1, a2, a3, a4, a5, a6, a7, a8;
    a1 = input[i];
    a2 = (i + BLOCK_SIZE) < len ? input[i + BLOCK_SIZE] : INFINITY;
    a3 = (i + 2 * BLOCK_SIZE) < len ? input[i + 2 * BLOCK_SIZE] : INFINITY;
    a4 = (i + 3 * BLOCK_SIZE) < len ? input[i + 3 * BLOCK_SIZE] : INFINITY;
    a5 = (i + 4 * BLOCK_SIZE) < len ? input[i + 4 * BLOCK_SIZE] : INFINITY;
    a6 = (i + 5 * BLOCK_SIZE) < len ? input[i + 5 * BLOCK_SIZE] : INFINITY;
    a7 = (i + 6 * BLOCK_SIZE) < len ? input[i + 6 * BLOCK_SIZE] : INFINITY;
    a8 = (i + 7 * BLOCK_SIZE) < len ? input[i + 7 * BLOCK_SIZE] : INFINITY;
    min_val = a1;
    min_idx = i;
    if (a2 < min_val) {
      min_val = a2;
      min_idx = i + BLOCK_SIZE;
    }
    if (a3 < min_val) {
      min_val = a3;
      min_idx = i + 2 * BLOCK_SIZE;
    }
    if (a4 < min_val) {
      min_val = a4;
      min_idx = i + 3 * BLOCK_SIZE;
    }
    if (a5 < min_val) {
      min_val = a5;
      min_idx = i + 4 * BLOCK_SIZE;
    }
    if (a6 < min_val) {
      min_val = a6;
      min_idx = i + 5 * BLOCK_SIZE;
    }
    if (a7 < min_val) {
      min_val = a7;
      min_idx = i + 6 * BLOCK_SIZE;
    }
    if (a8 < min_val) {
      min_val = a8;
      min_idx = i + 7 * BLOCK_SIZE;
    }
  }

  smem_val[tx] = min_val;
  smem_idx[tx] = min_idx;
  __syncthreads();

  // in-place reduction in shared memory
  if (blockDim.x >= 1024 && tx < 512 && smem_val[tx + 512] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 512];
    smem_idx[tx] = min_idx = smem_idx[tx + 512];
  }
  __syncthreads();

  if (blockDim.x >= 512 && tx < 256 && smem_val[tx + 256] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 256];
    smem_idx[tx] = min_idx = smem_idx[tx + 256];
  }
  __syncthreads();

  if (blockDim.x >= 256 && tx < 128 && smem_val[tx + 128] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 128];
    smem_idx[tx] = min_idx = smem_idx[tx + 128];
  }
  __syncthreads();

  if (blockDim.x >= 128 && tx < 64 && smem_val[tx + 64] < smem_val[tx]) {
    smem_val[tx] = min_val = smem_val[tx + 64];
    smem_idx[tx] = min_idx = smem_idx[tx + 64];
  }
  __syncthreads();

  // unrolling warp
  if (tx < 32) {
    volatile float *vsmem_val = smem_val;
    volatile int *vsmem_idx = smem_idx;
    if (vsmem_val[tx + 32] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 32];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 32];
    }
    if (vsmem_val[tx + 16] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 16];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 16];
    }
    if (vsmem_val[tx + 8] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 8];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 8];
    }
    if (vsmem_val[tx + 4] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 4];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 4];
    }
    if (vsmem_val[tx + 2] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 2];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 2];
    }
    if (vsmem_val[tx + 1] < vsmem_val[tx]) {
      vsmem_val[tx] = min_val = vsmem_val[tx + 1];
      vsmem_idx[tx] = min_idx = vsmem_idx[tx + 1];
    }
  }

  if (tx == 0) {
    output_val[blockIdx.x + offset_out] = min_val;
    output_idx[blockIdx.x + offset_out] = (input_idx == nullptr) ? min_idx : input_idx[min_idx];
  }
}

int main(int argc, char *argv[]) {
  if (argc < 2 || argc > 2) {
    cout << "Usage: " << argv[0] << " size\n";
    return -1;
  }
  const int len = 1L << (atoi(argv[1]));
  float *h_a = (float *)malloc(len * sizeof(float));

  if (h_a == nullptr) {
    cout << "Cannot allocate memory\n";
    exit(-1);
  }
  srand(0);
  clock_t begin = clock();
  for (int i = 0; i < len; ++i) {
    h_a[i] = rand() / (float)RAND_MAX;
  }
  cout << "Create dataset elapsed time: " << double(clock() - begin) / CLOCKS_PER_SEC * 1000
       << " ms\n";

  const int n_streams = 4;
  hipStream_t streams[n_streams];
  const int stream_size = ceil((float)len / (8 * BLOCK_SIZE * n_streams)) * (8 * BLOCK_SIZE);
  cout << "Stream size: " << stream_size << "\n";
  for (int i = 0; i < n_streams; ++i) {
    CHECK(hipStreamCreate(&streams[i]))
  }

  int len_out = ceil((float)stream_size / (BLOCK_SIZE * 8)) * n_streams;
  int len_out1 = ceil((float)len_out / (BLOCK_SIZE * 8));
  float *h_val = (float *)malloc(sizeof(float) * len_out1);
  int *h_idx = (int *)malloc(sizeof(int) * len_out1);

  float *d_a;
  float *d_val, *d_val1;
  int *d_idx, *d_idx1;

  CHECK(hipMalloc((void **)&d_a, sizeof(float) * len));
  CHECK(hipMalloc((void **)&d_val, sizeof(float) * len_out));
  CHECK(hipMalloc((void **)&d_idx, sizeof(int) * len_out));
  CHECK(hipMalloc((void **)&d_val1, sizeof(float) * len_out1));
  CHECK(hipMalloc((void **)&d_idx1, sizeof(int) * len_out1));

  CHECK(hipMemcpy(d_a, h_a, sizeof(float) * len, hipMemcpyHostToDevice));

  for (int i = 0; i < n_streams; ++i) {
    int offset = i * stream_size;
    int offset_out = i * (float)stream_size/(BLOCK_SIZE * 8);
    getMin<<<ceil((float)stream_size/(BLOCK_SIZE * 8)), BLOCK_SIZE, 0, streams[i]>>>(d_a, nullptr, offset, len, offset_out, d_val, d_idx);
  }

  CHECK(hipDeviceSynchronize());

  getMin<<<len_out1, BLOCK_SIZE>>>(d_val, d_idx, 0, len_out, 0, d_val1, d_idx1);

  CHECK(hipDeviceSynchronize());

  CHECK(hipMemcpy(h_val, d_val1, sizeof(float) * len_out1,
                   hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(h_idx, d_idx1, sizeof(int) * len_out1,
                   hipMemcpyDeviceToHost));



  float val = h_val[0];
  int idx = h_idx[0];
  for (int i = 0; i < len_out1; ++i) {
    if (h_val[i] < val) {
      val = h_val[i];
      idx = h_idx[i];
    }
  }

  cout << "Number of elements: " << len << ", min val: " << val
       << ", min idx: " << idx << "\n";

  for (int i = 0; i < n_streams; ++i) {
    CHECK(hipStreamDestroy(streams[i]));
  }

  // Free device
  CHECK(hipFree(d_a));
  CHECK(hipFree(d_val));
  CHECK(hipFree(d_idx));
  CHECK(hipFree(d_val1));
  CHECK(hipFree(d_idx1));

  // Free host
  free(h_a);
  free(h_val);
  free(h_idx);
  return 0;
}
